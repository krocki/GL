
#include <hip/hip_runtime.h>
/*
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

// Utilities and system includes

// clamp x to range [a, b]
__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ int clamp(int x, int a, int b)
{
    return max(a, min(b, x));
}

// convert floating point rgb color to 8-bit integer
__device__ int rgbToInt(float r, float g, float b)
{
    r = clamp(r, 0.0f, 255.0f);
    g = clamp(g, 0.0f, 255.0f);
    b = clamp(b, 0.0f, 255.0f);
    return (int(b)<<16) | (int(g)<<8) | int(r);
}

__global__ void cudainit(unsigned int *g_odata, int imgw) {

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

    g_odata[y*imgw+x] = (tx + ty + bw + bh ) % 7 == 0 ? 0xffffffff : 0x00000000;
}

__global__ void cudaProcess(unsigned int *g_odata, int imgw)
{
    extern __shared__ uchar4 sdata[];

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    int bw = blockDim.x;
    int bh = blockDim.y;
    int x = blockIdx.x*bw + tx;
    int y = blockIdx.y*bh + ty;

    unsigned int in = g_odata[y*imgw+x];
    unsigned int out;
    unsigned int n=0;

    // counting neighbors
    if (y > 0        && x > 0        ) n += (g_odata[(y-1)*imgw+(x-1)] > 0) ? 1 : 0;
    if (y > 0                        ) n += (g_odata[(y-1)*imgw+(x  )] > 0) ? 1 : 0;
    if (y > 0        && x < (imgw-1) ) n += (g_odata[(y-1)*imgw+(x+1)] > 0) ? 1 : 0;
    if (                x > 0        ) n += (g_odata[(y  )*imgw+(x-1)] > 0) ? 1 : 0;
    if (                x < (imgw-1) ) n += (g_odata[(y  )*imgw+(x+1)] > 0) ? 1 : 0;
    if (y < (imgw-1) && x > 0        ) n += (g_odata[(y+1)*imgw+(x-1)] > 0) ? 1 : 0;
    if (y < (imgw-1)                 ) n += (g_odata[(y+1)*imgw+(x  )] > 0) ? 1 : 0;
    if (y < (imgw-1) && x < (imgw-1) ) n += (g_odata[(y+1)*imgw+(x+1)] > 0) ? 1 : 0;

    // rules of game of life
    if (n < 2 || n > 3) out = 0;
    else if (n == 3 && g_odata[(y)*imgw+(x)] == 0) out = 0xffffffff;
    else out = in;

    g_odata[y*imgw+x] = out;//rgbToInt(c4.z, c4.y, c4.x);
}

extern "C" void launch_cudainit(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int imgw)
{
    cudainit<<< grid, block, sbytes >>>(g_odata, imgw);

}

extern "C" void launch_cudaProcess(dim3 grid, dim3 block, int sbytes, unsigned int *g_odata, int imgw)
{
    cudaProcess<<< grid, block, sbytes >>>(g_odata, imgw);

}
