
#include <hip/hip_runtime.h>
// kmrocki 1/15/19

__global__ void cudainit(unsigned int *canvas, int imgw) {

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    canvas[y*imgw+x] = (threadIdx.x + threadIdx.y + blockDim.x + blockDim.y ) % 7 == 0 ? 0xffffffff : 0x00000000;
}

__global__ void gameoflife(unsigned int *canvas, int imgw)
{
    // thread indices
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

    // original value at location y,x
    unsigned int in = canvas[y*imgw+x];
    unsigned int n=0; // number of neighbors

    // counting neighbors
    if (y > 0        && x > 0        ) n += (canvas[(y-1)*imgw+(x-1)] > 0) ? 1 : 0;
    if (y > 0                        ) n += (canvas[(y-1)*imgw+(x  )] > 0) ? 1 : 0;
    if (y > 0        && x < (imgw-1) ) n += (canvas[(y-1)*imgw+(x+1)] > 0) ? 1 : 0;
    if (                x > 0        ) n += (canvas[(y  )*imgw+(x-1)] > 0) ? 1 : 0;
    if (                x < (imgw-1) ) n += (canvas[(y  )*imgw+(x+1)] > 0) ? 1 : 0;
    if (y < (imgw-1) && x > 0        ) n += (canvas[(y+1)*imgw+(x-1)] > 0) ? 1 : 0;
    if (y < (imgw-1)                 ) n += (canvas[(y+1)*imgw+(x  )] > 0) ? 1 : 0;
    if (y < (imgw-1) && x < (imgw-1) ) n += (canvas[(y+1)*imgw+(x+1)] > 0) ? 1 : 0;

    // rules of game of life
    unsigned int out;
    if (n < 2 || n > 3) out = 0;
    else if (n == 3 && canvas[(y)*imgw+(x)] == 0) out = 0xffffffff;
    else out = in;

    canvas[y*imgw+x] = out;
}

extern "C" void launch_cudainit(dim3 grid, dim3 block, int sbytes, unsigned int *canvas, int imgw)
{
    cudainit<<< grid, block, sbytes >>>(canvas, imgw);

}

extern "C" void launch_process(dim3 grid, dim3 block, int sbytes, unsigned int *canvas, int imgw)
{
    gameoflife<<< grid, block, sbytes >>>(canvas, imgw);

}
